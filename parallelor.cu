#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
__global__ void BFShigh(int t,int *m,int *st,int *te,int *d,int *chan,int round,int edgesize,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=edgesize)return;
	int from=st[i];
	if (chan[from]<0)return;
	chan[from]=-1;
	int to=te[i];
	d[to]=round;
	if((to%nodenum)/(WD+1)==t)*m=1;
}
__global__ void initchan(int s,int *chan,int *d,int *pred,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nodenum*LY)return;
	int bi=i%nodenum;
	int W=WD+1;
	chan[i]=(bi/W==s)?1:-1;
	d[i]=(bi/W==s)?0:inf;
	pred[i]=d[i];
}
__global__ void chanchan(int *m,int *pred,int *d,int *chan,int nodenum)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=nodenum*LY)return;
	chan[i]=-1;
	if(d[i]<pred[i])
	{
		chan[i]=1;
		pred[i]=d[i];
	}
}
void parallelor::copydata(int s,vector<edge>&edges,int nodenum){
	memset(pre,-1,sizeof(int)*nodenum);
	*m=0;
	for(int i=0;i<nodenum;i++)
		d[i]=INT_MAX/2;
	d[s]=0;
	for(int i=0;i<edges.size();i++)
		aedges[i]=edges[i];
	hipMemcpy(dev_edges,aedges,edges.size()* sizeof(edge),hipMemcpyHostToDevice);
	hipMemcpy(dev_m,m,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,sizeof(int)*nodenum,hipMemcpyHostToDevice);
	hipMemcpy(dev_pre,pre,sizeof(int)*nodenum,hipMemcpyHostToDevice);
};
void parallelor::dellocate(){
	/*delete[]d;
	delete[]pre;
	delete[]aedges;
	delete m;
	hipFree(dev_edges);
	hipFree(dev_m);
	hipFree(dev_d);
	hipFree(dev_pre);*/
};
void parallelor::allocate(int maxn,int maxedge){
	m=new int;
	d=new int[maxn],pre=new int[maxn];
	aedges=new edge[maxedge];
	hipMalloc(&dev_edges, sizeof(edge)*maxedge);
	hipMalloc((void**)&dev_d,maxn*sizeof(int));
	hipMalloc((void**)&dev_pre,maxn*sizeof(int));
	hipMemcpy(duan,dev_duan,duansize*sizeof(int),hipMemcpyDeviceToHost);
	hipMalloc((void**)&dev_m,sizeof(int));
}
bool parallelor::cutcake(int index){

	cout<<"cut "<<index<<endl;
	if(maxbw-(index+1)*10>=0)
		maxbw-=(index+1)*10;
	else
		{
			cout<<"failure"<<endl;
			return false;
		}
	hleveln[index]++;
	return true;
};
void parallelor::topsort()
{
	cout<<" in top sort "<<endl;
	queue<int>zero;
	vector<int>order(nodenum*LY,-1);
	for(int i=0;i<nodenum*LY;i++)
		zero.push(i);
	int biao=0;
	while(!zero.empty())
	{
		int node=zero.front();
		zero.pop();
		order[node]=biao++;
		for(int i=0;i<neibn[node].size();i++)
		{
			if((--ancestor[neibn[node][i]])==0)
				zero.push(neibn[node][i]);
		}
	}
	vector<pair<int,int>>tmp;
	for(int i=0;i<order.size();i++)
		tmp.push_back(make_pair(i,order[i]));
	sort(tmp.begin(),tmp.end(),pairless());
	for(int i=0;i<order.size();i++)
		ordernode.push_back(tmp[i].first);
};
void parallelor::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	nodenum=ginf.enodesize;
	edges=extenedges;
	mark=new int;
	*mark=0;
	W=WD+1;
	int *d,*dev_d,*pred,*dev_pred;
	st=new int[2*WD*edges.size()*LY];
	te=new int[2*WD*edges.size()*LY];
	chan=new int[nodenum*LY];
	d=new int[nodenum*LY];
	pred=new int[nodenum*LY];
	vector<vector<int>>nein(nodenum*LY,vector<int>());
	vector<int>as(nodenum*LY,0);
	ancestor=as;
	neibn=nein;
	cout<<"gsdfs"<<endl;
	for(int k=0;k<LY;k++)
	{
		int startn=k*nodenum;
		for(int i=0;i<edges.size();i++)
			for(int j=0;j<W-1;j++)
			{
				int s=edges[i].s*W+j+startn;
				int t=edges[i].t*W+j+1+startn;
				ancestor[t]++;
				neibn[s].push_back(t);
				neibn[t].push_back(s);
			}
	}
	cout<<"before sort "<<endl;
	topsort();
	int count=0;
	cout<<"sort out "<<endl;
	for(int i=0;i<nodenum*LY;i++)
		for(int j=0;j<neibn[ordernode[i]].size();j++)
		{
			st[count]=ordernode[i];
			te[count]=neibn[ordernode[i]][j];
			count++;
		}
	cout<<"asdasd"<<endl;
	for(int i=0;i<nodenum*LY;i++)
	{
		chan[i]=-1;
		d[i]=INT_MAX/2;
		pred[i]=d[i];
	}
	cout<<"hrerr"<<endl;
	hipMalloc((void**)&dev_chan,nodenum*LY*sizeof(int));
	hipMalloc((void**)&dev_st,LY*WD*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,LY*WD*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_d,LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_pred,LY*nodenum*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMemcpy(dev_chan,chan,nodenum*LY*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_te,te,LY*WD*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,LY*WD*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_d,d,LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_pred,pred,LY*nodenum*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
	cout<<"get out"<<endl;
};

void parallelor::initprepush(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	maxbw=500;
	//allocate in cuda
	nodenum=ginf.enodesize;
	edges=extenedges;
	cout<<"out cuda init"<<endl;
}
parallelor::parallelor()
{

};
vector<int> parallelor:: routalg(int s,int t,int bw)
{
	cout<<"blasting "<<endl;
	int E=2*edges.size()*WD*LY;
	int kk=1;
	for(int i=0;i<1;i++)
	{
		*mark=0;
		initchan<< <(nodenum*LY/WORK_SIZE)+1, WORK_SIZE >> >(s,dev_chan,dev_d,dev_pred,nodenum);
		hipMemcpy(dev_m,&mark, sizeof(int), hipMemcpyHostToDevice);
		do{
			hipMemcpy(chan,dev_chan,nodenum*sizeof(int), hipMemcpyDeviceToHost);
			int cc=0;
			BFShigh << <(E/WORK_SIZE)+1, WORK_SIZE >> >(t,dev_m,dev_st,dev_te,dev_d,dev_chan,kk,E,nodenum);
			chanchan<< <(nodenum*LY/WORK_SIZE)+1, WORK_SIZE >> >(dev_m,dev_pred,dev_d,dev_chan,nodenum);
			hipMemcpy(mark, dev_m, sizeof(int), hipMemcpyDeviceToHost);
			kk++;
		}
		while(*mark==0);
		cout<<"out here is !"<<endl;
		//cout<<"kk is: "<<kk<<endl;
	}
	cout<<"out routalg"<<endl;
	return vector<int>();
};
int fls(int x)
{
	int position;
	int i;
	if(x!=0)
		for(i=(x>>1),position=0;i!=0;++position)
			i>>=1;
	else
		position=-1;
	return pow(2,position+1);
}