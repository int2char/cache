#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include"pathalg.h"
static const int WORK_SIZE =258;
parallelpush::parallelpush()
{
	cout<<"fuck c++,rubish!!!!"<<endl;
};
void parallelpush::init(vector<edge>&extenedges,vector<vector<int>>&relate,ginfo ginf){
	cout<<"in cuda init"<<endl;
	nodenum=ginf.enodesize;
	pnodesize=ginf.pnodesize;
	edges=extenedges;
	cout<<"out cuda init"<<endl;
	W=WD+1;
	h=new int[W*pnodesize*LY];
	v=new int[W*pnodesize*LY];
	mark=new int;
	vector<vector<int>>rawneie(pnodesize,vector<int>());
	for(int i=0;i<edges.size();i++)
		{
			int s=edges[i].s;
			int t=edges[i].t;
			rawneie[s].push_back(i+1);
			rawneie[t].push_back(-(i+1));
		}
	max=0;
	for(int i=0;i<rawneie.size();i++)
		if(rawneie[i].size()>max)max=rawneie[i].size();
	max++;
	cout<<"max is: "<<max<<endl;
	neie=new int[pnodesize*max];
	for(int i=0;i<pnodesize;i++)
		{
			for(int j=0;j<max;j++)
			{
				if(j<rawneie[i].size())
					neie[i*max+j]=rawneie[i][j];
				else
					neie[i*max+j]=INT_MAX;
			}
		}
	emark=new int[LY*edges.size()];
	esign=new int[LY*edges.size()];
	st=new int[edges.size()];
	te=new int[edges.size()];

	for(int i=0;i<edges.size();i++)
		{
			st[i]=edges[i].s*W;
			te[i]=edges[i].t*W;
		}

	hipMalloc((void**)&dev_h,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_mark,sizeof(int));
	hipMalloc((void**)&dev_v,LY*W*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_neie,max*pnodesize*sizeof(int));
	hipMalloc((void**)&dev_esign,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_emark,LY*edges.size()*sizeof(int));
	hipMalloc((void**)&dev_st,edges.size()*sizeof(int));
	hipMalloc((void**)&dev_te,edges.size()*sizeof(int));
	hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_neie,neie,max*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_st,st,edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_te,te,edges.size()*sizeof(int),hipMemcpyHostToDevice);
};
__global__ void push(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]>INT_MAX/2&&h==hnbj+1)
				{
					dev_emark[eid]=(ebj>0)?nbj:i;
					value--;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void push1(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	*mark=1;
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]==0&&h==hnbj+1)
				{
					dev_emark[eid]++;
					value--;
					*mark=1;
				}
				minheight=(minheight<hnbj)?minheight:hnbj;
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void push2(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*st,int*te,int*neie,int N,int W,int E,int s,int t,int*mark,int max)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	int node=bi/W;
	if(i>=N*LY||value==0||node==s||node==t)return;
	int ly=i/N;
	int off=i%W;
	int h=dev_h[i];
	int b=node*max;
	int minheight=INT_MAX;
	int ebj,nbj,hnbj,eid,seid;
	for(int j=0;j<max;j++)
	{
		ebj=neie[b+j];
		if(value>0&&ebj<INT_MAX){
			seid=abs(ebj)-1;
			eid=ly*E+seid;
			nbj=-1;
			bool btest=(ebj^dev_esign[eid])>0;
			//bool b1=ebj>0&&dev_esign[eid]>0;
			//bool b2=ebj<0&&dev_esign[eid]<0&&abs(dev_esign[eid])==off;
			if(btest)
			{
				if(ebj>0&&off<W-1)
					nbj=te[seid]+off+1;
				if(ebj<0&&off>0)
					nbj=st[seid]+off-1;
				if(nbj<0)continue;
				nbj+=ly*N;
				hnbj=dev_h[nbj];
				if(dev_emark[eid]==0&&h==hnbj+1)
				{
					dev_emark[eid]++;
					value--;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX){dev_h[i]=minheight+1;*mark=1;}
};
__global__ void aggregate3(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E,int N)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
		int s,t;
	int bottom=(i/E)*N;
	int bi=i%E;
	if(dev_emark[i]>0)
	{
		if(dev_esign[i]>0)
		{
			s=dev_st[bi];
			t=dev_te[bi]+1;
		}
		else
		{
			t=dev_st[bi];
			s=dev_te[bi]+1;
		}
		s+=bottom;
		t+=bottom;
		for(int k=0;k<W;k++)
			{
				int h1=dev_h[s+k];
				int h2=dev_h[t+k];
				if(dev_v[s+k]>0&&h1==h2+1)
				{
					atomicSub(&dev_v[s+k],1);
					atomicAdd(&dev_v[t+k],1);
					//dev_esign[i]=(dev_esign[i]>0)?-(k+t)%W:1;
					dev_esign[i]*=-1;
					break;
				}
			}
	}

	dev_emark[i]=0;
};
__global__ void pushrelable(int*dev_h,int*dev_v,int* dev_esign,int* dev_emark,int*dev_neie,int*dev_nein,int N,int max,int W,int s,int t,int*mark)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	int value=dev_v[i];
	if(i>=N*LY||value==0||bi/W==s||bi/W==t)return;
	int h=dev_h[i];
	int b=i*max;
	int minheight=INT_MAX;
	for(int j=0;j<max;j++)
	{
		int nbj=dev_nein[b+j];
		if(value>0&&nbj<INT_MAX)
		{
			int ebj=dev_neie[b+j];
			int hnbj=dev_h[nbj];
			int eid=abs(ebj)-1;
			if((ebj^dev_esign[eid])>0)
			{
				if(dev_emark[eid]==i)
				{
					dev_emark[eid]++;
					atomicSub(&dev_v[i],1);
					atomicAdd(&dev_v[nbj],1);
					value--;
					dev_esign[eid]*=-1;
					*mark=1;
				}
				minheight=min(minheight,hnbj);
			}
		}
		else
			break;
	}
	if(value>0&&minheight<INT_MAX/2){dev_h[i]=minheight+1;*mark=1;}
};

__global__ void aggregate4(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
	int s,t;
	dev_emark[i]=INT_MAX;
	if(dev_esign[i]>0)
	{
		s=dev_st[i];
		t=dev_te[i]+1;
	}
	else
	{
		t=dev_st[i];
		s=dev_te[i]+1;
	}
	for(int k=0;k<W;k++)
		{
			int h1=dev_h[s+k];
			int h2=dev_h[t+k];
			if(dev_v[s+k]>0&&h1==h2+1)
			{
				dev_emark[i]=s+k;
				break;
			}
		}
};
__global__ void aggregate2(int* dev_esign,int*dev_v,int* dev_emark,int W,int E)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i>=E*LY)return;
	int emid=dev_emark[i];
	if(emid<INT_MAX)
	{
		int s=abs(dev_esign[i])-2+emid%W;
		if(dev_esign[i]>0)
		{	atomicSub(&dev_v[s],1);
			atomicAdd(&dev_v[emid],1);
		}
		else
		{	atomicAdd(&dev_v[s],1);
			atomicSub(&dev_v[emid],1);
		}
		dev_esign[i]*=-1;
	}
	dev_emark[i]=INT_MAX;
};
__global__ void aggregate5(int* dev_esign,int* dev_v,int* dev_emark,int* dev_st,int* dev_te,int*dev_h,int W,int E)
{
        int i = threadIdx.x + blockIdx.x*blockDim.x;
        if(i>=E*LY*W)return;
        int s,t;
        int eid=i/W;
        int k=i%W;
        if(dev_esign[eid]>0)
        {
                s=dev_st[eid];
                t=dev_te[eid]+1;
        }
        else
        {
                t=dev_st[eid];
                s=dev_te[eid]+1;
        }
        int h1=dev_h[s+k];
        int h2=dev_h[t+k];
        if(dev_v[s+k]>0&&h1==h2+1)
                dev_emark[eid]=s+k;
};

__global__ void relable(int*dev_h,int*dev_v,int N,int*mark,int*dev_nein,int*dev_neie,int *dev_esign,int max,int W,int s,int t)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	int bi=i%N;
	if(i>=N*LY||dev_v[i]==0||bi/W==s||bi/W==t)return;
	int b=i*max;
	int mini=INT_MAX;
	for(int j=0;j<max;j++)
	{
		if(dev_nein[b+j]<INT_MAX)
		{
			if((dev_neie[b+j]^dev_esign[abs(dev_neie[b+j])-1])>0)
				mini=min(mini,dev_h[dev_nein[b+j]]);
		}
		else
			break;
	}
	if(mini!=INT_MAX)
		dev_h[i]=mini+1,*mark=1;
};

pair<int,int> parallelpush::prepush(int s,int t,int bw)
{
	cout<<"**********************************"<<endl;
	cout<<"parral: "<<LY<<" "<<pnodesize<<" "<<s<<" "<<t<<endl;
	time_t start,end;
	start=clock();
	for(int i=0;i<LY*edges.size();i++)
		emark[i]=0;
	for(int k=0;k<LY;k++)
		for(int i=0;i<edges.size();i++)
			esign[i+k*edges.size()]=1;
	for(int i=0;i<W*LY*pnodesize;i++)
		{
			h[i]=0;
			v[i]=0;
		}
	for(int k=0;k<LY;k++)
		{
		for(int i=0;i<edges.size();i++)
			if(edges[i].s==s)
				{
				v[k*W*pnodesize+W*edges[i].t+1]=1;
				esign[k*edges.size()+i]*=-1;
				}
		}
	for(int k=0;k<LY;k++)
		{
		int start=k*W*pnodesize;
		for(int i=s*W;i<s*W+W;i++)
			h[i+start]=WD;
		}
	hipMemcpy(dev_h,h,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_v,v,LY*W*pnodesize*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_esign,esign,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(dev_emark,emark,LY*edges.size()*sizeof(int),hipMemcpyHostToDevice);
	*mark=1;
	int time=0;
	cout<<"max is "<<max<<endl;
	while(*mark!=0)
	{
		if(time%100==0)
			{*mark=0;
			hipMemcpy(dev_mark,mark,sizeof(int),hipMemcpyHostToDevice);}
		push2<<<LY*W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,dev_esign,dev_emark,dev_st,dev_te,dev_neie,W*pnodesize,W,edges.size(),s,t,dev_mark,max);
		//push1<<<LY*W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,dev_esign,dev_emark,dev_neie,dev_nein,W*pnodesize,max,W,s,t,dev_mark);
		//aggregate2<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W,edges.size());
		aggregate3<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,dev_st,dev_te,dev_h,W,edges.size(),W*pnodesize);
		/*hipMemcpy(emark,dev_emark,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
		for(int i=0;i<LY*edges.size();i++)
			if(emark[i]>0)
				cout<<"gota... "<<i<<"s:"<<st[i]<<" "<<te[i]<<" "<<emark[i]<<endl;*/
		//relable<<<LY*W*pnodesize/WORK_SIZE+1,WORK_SIZE>>>(dev_h,dev_v,W*pnodesize,dev_mark,dev_nein,dev_neie,dev_esign,max,W,s,t);
		//aggregate2<<<LY*edges.size()/WORK_SIZE+1,WORK_SIZE>>>(dev_esign,dev_v,dev_emark,W,edges.size(),W*pnodesize,dev_mark);
		if(time%100==0)
			hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
		/*hipMemcpy(v,dev_v,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(h,dev_h,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
		hipMemcpy(esign,dev_esign,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
		int flow=0;
		cout<<"************* "<<time<<endl;
		for(int i=0;i<LY*W*pnodesize;i++)
			if(v[i]!=0)
				{
					int bi=i%(W*pnodesize);
					if(bi/W==t)flow+=v[i];
					cout<<i/(W*pnodesize)<<"\t"<<bi<<"\t"<<bi/W<<"\t"<<bi%W<<"\t"<<h[i]<<"\t"<<v[i]<<endl;
					/*if(i==319)
					{
						for(int j=0;j<max;j++)
							if(nein[i*max+j]<INT_MAX)
								cout<<neie[i*max+j]<<" "<<esign[abs(neie[i*max+j])-1]<<" "<<h[nein[i*max+j]]<<endl;
					}*/
				//}
		//cout<<"mark "<<*mark<<endl;
		time++;
	}
	hipMemcpy(mark,dev_mark,sizeof(int),hipMemcpyDeviceToHost);
	end=clock();
	cout<<"GPU time is: "<<end-start<<endl;
	hipMemcpy(v,dev_v,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(h,dev_h,LY*W*pnodesize*sizeof(int),hipMemcpyDeviceToHost);
	int flow=0;
	for(int i=0;i<LY*W*pnodesize;i++)
		if(v[i]!=0)
			{
				int bi=i%(W*pnodesize);
				if(bi/W==t)flow+=v[i];
				//cout<<i/(W*pnodesize)<<" "<<bi<<" "<<bi/W<<" "<<bi%W<<" "<<h[i]<<" "<<v[i]<<endl;
			}
	hipMemcpy(esign,dev_esign,LY*edges.size()*sizeof(int),hipMemcpyDeviceToHost);
	int count=0;
	for(int i=0;i<edges.size()*LY;i++)
		if(esign[i]<0)
			count++;
	cout<<"resort"<<endl;
	/*for(int i=0;i<edges.size();i++)
		{
			if(esign[i]<0)
			{
				int sorce=edges[i].t*W;
				if(sorce/W==t)
				{
					int pre=edges[i].s*W;
					cout<<pre<<" ";
					while((pre/W)!=s)
					{
						int flag=0;
						for(int h=0;h<W;h++)
						{
							pre++;
							for(int k=0;k<max;k++)
								{
									if(nein[pre*max+k]<INT_MAX)
										if(esign[abs(neie[pre*max+k])-1]<0&&neie[pre*max+k]<0)
										{
											esign[abs(neie[pre*max+k])-1]*=-1;
											pre=edges[abs(neie[pre*max+k])-1].s*W;
											cout<<pre<<" ";
											flag=1;
										}
										if(flag==1)break;
								}
							if(flag==1)break;
						}
					}
					cout<<endl;
				}
			}
		}*/
	cout<<"flow is"<<flow<<endl;
	cout<<"count is "<<count<<endl;
	cout<<"die is "<<time<<endl;
	return make_pair(flow,end-start);
};
void parallelpush:: dellocate()
{
	/*delete[] h;
	delete[] minarray;
	delete[] v;
	delete[] mark;
	delete[] neie;
	delete[] nein;
	delete[]emark;
	delete[]esign;*/
	hipFree(dev_h);
	hipFree(dev_mark);
	hipFree(dev_v);
	hipFree(dev_neie);
	hipFree(dev_nein);
	hipFree(dev_esign);
	hipFree(dev_emark);
}
parallelpush:: ~parallelpush(){dellocate();};



